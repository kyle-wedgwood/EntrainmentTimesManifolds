#include "hip/hip_runtime.h"
/* Driver for use of finder operator to find isochrons
   Currently only works for planar models */
#include <iostream>
#include <cstdlib>
#include <cmath>
#include "EntrainmentTimeFinder.hpp"
#include "parameters.hpp"

void SetDayLengthPars( const double day_length,
                       double& host_length_scaling,
                       double& host_length_shift)
{
  host_length_scaling = sqrt(pow(sin(pi/12.0*day_length),
        2)/(2.0*(1.0-cos(pi/12.0*day_length))));

  if (((day_length < 12) & (host_length_scaling > 0)) |
      ((day_length > 12) & (host_length_scaling < 0)))
  {
    host_length_scaling *= -1;
  }
  host_length_shift = 12.0/pi*asin(host_length_scaling);

  std::cout << host_length_scaling << std::endl;
  std::cout << host_length_shift << std::endl;
}

int main(int argc, char* argv[])
{

  double host_I = 50.0;
  double host_tShift = 0.0;
  double host_taux = 24.2;
  double day_length = 4.0; // in hours

  double dI = 10.0;
  double dtShift = 0.0;
  double dtaux = 0.0;
  double dday_length = 4.0;
  double host_length_scaling, host_length_shift;

  int npts = 5;
  char filename[80];

  EntrainmentTimeFinder* p_finder = new EntrainmentTimeFinder();

  p_finder->SetDimensions(make_int2(1024, 1024));
  p_finder->SetGeometry(make_double2(-2.0, -2.0),
                         make_double2(2.0, 2.0));
  p_finder->SetFinalTime(20000.0);

  for (int i=0; i<npts; i++)
  {
    // Load reference orbit
    sprintf(filename, "orbits/stable_orbit_FS_I_%d_N_%d_taux_%.1f.dat", (int)
        (host_I), (int) day_length, host_taux);
    p_finder->LoadOrbit(filename);
    SetDayLengthPars(day_length, host_length_scaling, host_length_shift);

    // Copy parameters to device
    hipMemcpyToSymbol(HIP_SYMBOL(I), &host_I, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(taux), &host_taux, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(tShift), &host_tShift, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(length_scaling), &host_length_scaling, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(length_shift), &host_length_shift, sizeof(double));

    // Find the entrainment times
    p_finder->FindEntrainmentTimes();

    // Save the data
    sprintf(filename, "results/EntrainmentTimes_FS_I_%d_N_%d_taux_%.1f.dat",
        (int) host_I, (int) day_length, host_taux);

    p_finder->SaveData(filename);

    // Update for next loop
    host_I += dI;
    host_tShift += dtShift;
    host_taux += dtaux;
    day_length += dday_length;

    std::cout << "Done " << i+1 << " of " << npts << std::endl;
  }

  delete(p_finder);

  return 0;
}
