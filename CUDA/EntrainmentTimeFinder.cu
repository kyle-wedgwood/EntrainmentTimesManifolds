#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <sstream>
#include <assert.h>
#include <vector>
#include "cu_error_functions.hpp"
#include "parameters.hpp"
#include "EntrainmentTimeFinder.hpp"
#include "CUDAKernels.hpp"

using namespace std;

EntrainmentTimeFinder::~EntrainmentTimeFinder()
{
  CUDA_CALL( hipFree( mpDev_result));
  CUDA_CALL( hipFree( mpDev_xMeshPts));
  CUDA_CALL( hipFree( mpDev_yMeshPts));
  CUDA_CALL( hipFree( mpDev_refOrbit));

  free( mpHost_result);
  free( mpHost_xMeshPts);
  free( mpHost_yMeshPts);
}

void EntrainmentTimeFinder::FindEntrainmentTimes()
{

  // Create meshes
  if (CreateMesh() > 0)
  {
    cout << "Simulation aborted." << endl;
    return;
  }

  // Transfer mesh data
  CUDA_CALL( hipMemcpy( mpDev_xMeshPts, mpHost_xMeshPts,
        mDim.x*sizeof(double), hipMemcpyHostToDevice));
  CUDA_CALL( hipMemcpy( mpDev_yMeshPts, mpHost_yMeshPts,
        mDim.y*sizeof(double), hipMemcpyHostToDevice));

  cout << "Meshes copied to device." << endl;

  // Reset memory
  CUDA_CALL( hipMemset( mpDev_result, 0.0, mDim.x*mDim.y*sizeof(double)));

  // Actually run the network
  cout << "Starting simulation..." << endl;

  FindEntrainmentTimesKernel<<<mNoBlocks,mNoThreads>>>( mDim,
                                                        mpDev_xMeshPts,
                                                        mpDev_yMeshPts,
                                                        mFinalTime,
                                                        mpDev_refOrbit,
                                                        mpDev_result);
  /*
  FindEntrainmentTimesPhaseKernel<<<mNoBlocks,mNoThreads>>>( mDim,
                                                             mpDev_xMeshPts,
                                                             mpDev_yMeshPts,
                                                             mFinalTime,
                                                             mpDev_result);
                                                             */
  CUDA_CHECK_ERROR();
  CUDA_CALL( hipDeviceSynchronize());

  cout << "Fourier averages computed successfully." << endl;

  // Copy data back
  CUDA_CALL( hipMemcpy( mpHost_result, mpDev_result,
        mDim.x*mDim.y*sizeof(double), hipMemcpyDeviceToHost));

  cout << "Data copied to host." << endl;
}

void EntrainmentTimeFinder::FindInsideFlag()
{
  // Create meshes
  if (CreateMesh() > 0)
  {
    cout << "Simulation aborted." << endl;
    return;
  }

  // Transfer mesh data
  CUDA_CALL( hipMemcpy( mpDev_xMeshPts, mpHost_xMeshPts,
        mDim.x*sizeof(double), hipMemcpyHostToDevice));
  CUDA_CALL( hipMemcpy( mpDev_yMeshPts, mpHost_yMeshPts,
        mDim.y*sizeof(double), hipMemcpyHostToDevice));

  cout << "Meshes copied to device." << endl;

  // Reset memory
  CUDA_CALL( hipMemset( mpDev_result, 0.0, mDim.x*mDim.y*sizeof(double)));

  // Actually run the network
  cout << "Starting simulation..." << endl;
  FindInsideFlagKernel<<<mNoBlocks,mNoThreads>>>( mDim,
                                                  mpDev_xMeshPts,
                                                  mpDev_yMeshPts,
                                                  mFinalTime,
                                                  mpDev_refOrbitAmp,
                                                  mpDev_result);

  CUDA_CHECK_ERROR();
  CUDA_CALL( hipDeviceSynchronize());

  cout << "Fourier averages computed successfully." << endl;

  // Copy data back
  CUDA_CALL( hipMemcpy( mpHost_result, mpDev_result,
        mDim.x*mDim.y*sizeof(double), hipMemcpyDeviceToHost));

  cout << "Data copied to host." << endl;
}

void EntrainmentTimeFinder::LoadOrbit( const char* filename)
{
  ifstream file;
  string line;
  std::cout << "Loading orbit from " << filename << std::endl;

  file.open( filename);
  int no_pts = (int)( 24.0/timestep);

  double temp_x;
  double temp_y;

  vector<double> x;
  vector<double> y;

  double2 p_hostRefOrbit[no_pts];

  while( getline( file, line))
  {
    istringstream iss( line);
    iss >> temp_x >> temp_y;
    x.push_back( temp_x);
    y.push_back( temp_y);
  }

  assert( x.size() == no_pts);

  // Put orbit into double2 array for easy copying
  for (int i=0; i<no_pts; i++)
  {
    p_hostRefOrbit[i].x = x[i];
    p_hostRefOrbit[i].y = y[i];
  }

  CUDA_CALL( hipMalloc( &mpDev_refOrbit, no_pts*sizeof( double2)));
  CUDA_CALL( hipMemcpy( mpDev_refOrbit, p_hostRefOrbit,
        no_pts*sizeof( double2), hipMemcpyHostToDevice));

  cout << "Reference orbit loaded." << endl;
}

void EntrainmentTimeFinder::LoadOrbitAmp( const char* filename)
{
  ifstream file;
  string line;
  file.open( filename);

  int no_pts = (int) ( 2*pi/theta_step) + 1;
  double temp_theta;
  double temp_rho;
  double p_hostRefOrbit[no_pts];

  vector<double> rho;

  while( getline( file, line))
  {
    istringstream iss( line);
    iss >> temp_theta >> temp_rho;
    rho.push_back( temp_rho);
  }

  assert( rho.size() == no_pts);

  // Put orbit into double array for easy copying
  for (int i=0; i<no_pts; i++)
  {
    p_hostRefOrbit[i] = rho[i];
  }

  CUDA_CALL( hipMalloc( &mpDev_refOrbitAmp, no_pts*sizeof( double)));
  CUDA_CALL( hipMemcpy( mpDev_refOrbitAmp, p_hostRefOrbit,
        no_pts*sizeof( double), hipMemcpyHostToDevice));

  cout << "Reference orbit loaded." << endl;
}

void EntrainmentTimeFinder::SetDimensions( int2 dim)
{
  mDim = dim;
  mDimFlag = true;
}

void EntrainmentTimeFinder::SetGeometry( double2 xMin, double2 xMax)
{
  mXMin = xMin;
  mXMax = xMax;
  mGeometryFlag = true;
}

void EntrainmentTimeFinder::SetFinalTime( double finalTime)
{
  mFinalTime = finalTime;
  mTimeFlag = true;
}

int EntrainmentTimeFinder::CreateMesh()
{
  if (!mDimFlag)
  {
    cout << "Dimensions not set. Aborting..." << endl;
    return 1;
  }
  if (!mGeometryFlag)
  {
    cout << "Geometry not set. Aborting..." << endl;
    return 2;
  }
  if (!mTimeFlag)
  {
    cout << "Simulation time not set. Aborting..." << endl;
  }

  // Allocate memory
  mpHost_xMeshPts = (double*) malloc( mDim.x*sizeof(double));
  mpHost_yMeshPts = (double*) malloc( mDim.y*sizeof(double));
  mpHost_result = (double*) malloc( mDim.x*mDim.y*sizeof(double));

  CUDA_CALL( hipMalloc( &mpDev_xMeshPts, mDim.x*sizeof(double)));
  CUDA_CALL( hipMalloc( &mpDev_yMeshPts, mDim.y*sizeof(double)));
  CUDA_CALL( hipMalloc( &mpDev_result, mDim.x*mDim.y*sizeof(double)));

  double dx = (mXMax.x-mXMin.x)/(mDim.x-1);
  double dy = (mXMax.y-mXMin.y)/(mDim.y-1);
  for (int i=0;i<mDim.x;++i)
  {
    mpHost_xMeshPts[i] = mXMin.x+i*dx;
  }
  for (int i=0;i<mDim.y;++i)
  {
    mpHost_yMeshPts[i] = mXMin.y+i*dy;
  }

  mNoThreads  = 512;
  mNoBlocks = (mDim.x*mDim.y-1)/mNoThreads;

  cout << "Created mesh object with "
       << mDim.x
       << " x "
       << mDim.y
       << " points."
       << endl;

  return 0;
}

void EntrainmentTimeFinder::SaveData( const char* filename)
{
  ofstream file;
  file.open( filename);
  file << 0.0 << "\t";
  for (int i=0;i<mDim.x;++i)
  {
    file << mpHost_xMeshPts[i] << "\t"; // x mesh points
  }
  file << endl;

  for (int j=0;j<mDim.y;++j)
  {
    file << mpHost_yMeshPts[j] << "\t"; // y mesh points
    for (int i=0;i<mDim.x;++i)
    {
      file << mpHost_result[j*mDim.y+i] << "\t"; // Actual Fourier averages
    }
    file << endl;
  }
  file.close();
  cout << "Entrainment times saved to " << filename << endl;
}
