#include "hip/hip_runtime.h"
#include <cstdlib>
#include "RungeKuttaSolver.hpp"
#include "NonlinearProblem.hpp"
#include "parameters.hpp"

// HELPER FUNCTIONS FOR RUNGE-KUTTA CLASS
#define _add(a,b) AddDouble2(a,b)
#define _scale(a,b) ScaleDouble2(a,b)

__device__ double2 AddDouble2( double2 a, double2 b)
{
  a.x += b.x;
  a.y += b.y;
  return a;
}

__device__ double2 ScaleDouble2( double a, double2 b)
{
  b.x *= a;
  b.y *= a;
  return b;
}

__device__ RungeKuttaSolver::RungeKuttaSolver( double dt, NonlinearProblem* pProblem)
{
  mDt = dt;
  mpProblem = pProblem;
}

__device__ void RungeKuttaSolver::RungeKuttaStep( double t, double2& u)
{
  double2 f0, f1;

  // Take predictive step
  mpProblem->ComputeF( t, u, f0);

  // Take corrective step
  mpProblem->ComputeF( t+mDt, _add(u,_scale(mDt,f0)), f1);

  // Perform full step
  u = _add( u, _scale(mDt/2.0,_add(f0,f1)));

}
