#include "hip/hip_runtime.h"
#include "CUDAKernels.hpp"
#include "RungeKuttaSolver.hpp"
#include "Kronauer.hpp"
#include "parameters.hpp"
#define norm(u,v) ((u.x-v.x)*(u.x-v.x) + (u.y-v.y)*(u.y-v.y))

__global__ void FindEntrainmentTimesKernel( const int2 dim,
                                            const double* pXMeshPts,
                                            const double* pYMeshPts,
                                            const double t_final,
                                            const double2* pRefOrbit,
                                            double* pResult)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index<dim.x*dim.y)
  {
    Kronauer* p_problem = new Kronauer();
    RungeKuttaSolver* p_solver = new RungeKuttaSolver( timestep, p_problem);

    // Initialise system
    int orbit_ind;
    double time = 0.0;
    double2 u;
    double2 ref_pt = pRefOrbit[0];
    u.x = pXMeshPts[index % dim.x];
    u.y = pYMeshPts[index / dim.x];

    do
    {
      time += timestep;

      p_solver->RungeKuttaStep( time, u);
      p_solver->RungeKuttaStep( time, ref_pt);

      //orbit_ind = (int) ( (time+24.0-tShift)/timestep) % (int) (24.0/timestep);
      //ref_pt = pRefOrbit[orbit_ind];

    } while ( (time<t_final) & (norm( u, ref_pt) > entrain_thresh*entrain_thresh));

    pResult[index] = time;

    delete( p_solver);
    delete( p_problem);
  }
}

__global__ void FindEntrainmentTimesPhaseKernel( const int2 dim,
                                                 const double* pXMeshPts,
                                                 const double* pYMeshPts,
                                                 const double t_final,
                                                 double* pResult)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index<dim.x*dim.y)
  {
    Kronauer* p_problem = new Kronauer();
    RungeKuttaSolver* p_solver = new RungeKuttaSolver( timestep, p_problem);

    // Initialise system
    double time = 0.0;
    double2 u;
    double2 u_old;
    u.x = pXMeshPts[index % dim.x];
    u.y = pYMeshPts[index / dim.x];

    do
    {
      time += timestep;

      u_old = u;
      p_solver->RungeKuttaStep( time, u);

    } while ( (time<t_final) & (( abs( fmod( time-tShift-24.0, 24.0) - entrained_phase) > 0.01) || (u.x > 0.0) || (u_old.x < 0.0)));

    pResult[index] = time;

    delete( p_solver);
    delete( p_problem);
  }
}

__global__ void FindInsideFlagKernel( const int2 dim,
                                      const double* pXMeshPts,
                                      const double* pYMeshPts,
                                      const double t_final,
                                      const double* pRefOrbitAmp,
                                      double* pResult)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index<dim.x*dim.y)
  {
    Kronauer* p_problem = new Kronauer();
    RungeKuttaSolver* p_solver = new RungeKuttaSolver( timestep, p_problem);

    // Initialise system
    int orbit_ind;
    double time = 0.0;
    double2 u;
    bool inside_flag = false;
    u.x = pXMeshPts[index % dim.x];
    u.y = pYMeshPts[index / dim.x];

    do
    {
      time += timestep;

      p_solver->RungeKuttaStep( time, u);

      orbit_ind = (int) ( ( atan2( u.y, u.x)+pi)/theta_step);
      inside_flag = ( sqrtf( u.x*u.x+u.y*u.y) < pRefOrbitAmp[orbit_ind]);

    } while ( (time<t_final) && (!inside_flag));

    pResult[index] = (double) inside_flag;

    delete( p_solver);
    delete( p_problem);
  }
}
