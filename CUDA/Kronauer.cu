#include "hip/hip_runtime.h"
#include "Kronauer.hpp"
#include "parameters.hpp"

__constant__ double I;
__constant__ double tShift;
__constant__ double taux;
__constant__ double length_scaling;
__constant__ double length_shift;

__device__ void Kronauer::ComputeF( double t, double2 u, double2& f)
{

  bool ft = (length_scaling+sin( 2.0*(pi/24.0)*(t-tShift-length_shift)) > 0);
  double alpha = alpha_0*sqrt(I/I_0);
  double B = G*alpha*ft*beta/(alpha+beta)*(1-0.4*u.y)*(1-0.4*u.x);

  f.x =
    (pi/12.0)*(mu*(u.x-(4.0/3.0)*u.x*u.x*u.x)-u.y*( (24.0/(0.99669*taux))*(24.0/(0.99669*taux))+k*B) );
  f.y = (pi/12.0)*(u.x+B);
}
